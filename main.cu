
#include <hip/hip_runtime.h>
#include <stdio.h>
#define NX 8
#define NY 8
#define DX (1./(float)NX)
#define DY (1./(float)NY)
#define N_ITERATIONS 8
#define N_THREADS 512
#define N_BLOCKS (NX*NY+N_THREADS+1)/N_THREADS


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Solves the Poisson equation via the Jacobi Method
// $$\nabla^2 \phi = f$$

__global__ void iteratePoisson(float* d_source, float* d_V1, float* d_V2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int n = j * NX + i;
    if ((i>0) && (i < NX-1) && (j>0) && (j < NY-1)) { //TODO: see what can be done about boundaries
        int n_top = (j-1) * NX + i;
        int n_bot = (j+1) * NX + i;
        int n_left = j * NX + (i-1);
        int n_right = j * NX + (i+1);
        //TODO: rewrite above in terms of n?

        d_V1[n] = 0.25f * (d_V2[n_top] + d_V2[n_bot] + d_V2[n_left] + d_V2[n_right]) +\
            d_source[n] * DX * DY;
            //TODO: check above for consistency. Does this need factor of 4?
    }
}

int main()
{
    float *h_source = (float *)malloc(NX*NY*sizeof(float));
    float *h_V = (float *)malloc(NX*NY*sizeof(float));

    float top_bc = 1;
    float bottom_bc = -1;
    float left_bc = 1;
    float right_bc = -1;

    float x;
    float y;

    for(int j =0; j<NY; j++){
        for (int i = 0; i < NX; i++){
            int n = NX*j + i;

            x = i*DX - NX/2 * DX;
            y = j*DY - NY/2 * DY;
            h_source[n] = x*x+y*y;  //TODO: set up source term

            if (j == 0){ // top row
                h_V[n] = top_bc;
            }
            else if (j==NY-1){ //bottom row
                h_V[n] = bottom_bc;
            }
            else if (i==0){ //left column
                h_V[n] = left_bc;
            }
            else if (i==NX-1){
                h_V[n] = right_bc;
            }
        }
    }


    float *d_source;
    float *d_V1;
    float *d_V2;

    //allocate GPU memory
    gpuErrchk(hipMalloc(&d_source, NX*NY*sizeof(float)));
    gpuErrchk(hipMalloc(&d_V1, NX*NY*sizeof(float)));
    gpuErrchk(hipMalloc(&d_V2, NX*NY*sizeof(float)));

    // copy V1, V2 from host to device
    gpuErrchk(hipMemcpy(d_source, h_source, NX*NY*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_V1, h_V, NX*NY*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_V2, h_V, NX*NY*sizeof(float), hipMemcpyHostToDevice));
    // gpuErrchk(cudaPeekAtLastError());

    printf("Blocks: %d\nThreads: %d\n", N_BLOCKS, N_THREADS);
    printf("Iteration %5d", 0);
    for (int i = 0; i < N_ITERATIONS; i += 2)
    {
        printf("\rIteration %5d", i);
        iteratePoisson<<<N_BLOCKS, N_THREADS>>>(d_source, d_V1, d_V2);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        iteratePoisson<<<N_BLOCKS, N_THREADS>>>(d_source, d_V2, d_V1);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }

    //copy V2 from device to host as final value
    gpuErrchk(hipMemcpy(h_source, d_source, NX*NY*sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_V, d_V2, NX*NY*sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_source));
    gpuErrchk(hipFree(d_V1));
    gpuErrchk(hipFree(d_V2));

    FILE* file_V1 = fopen("V1.dat", "w");
    FILE* file_source = fopen("source.dat", "w");

    for(int j =0; j<NY; j++){
        for (int i = 0; i < NX; i++){
            int n = NX*j + i;

            x = i*DX;
            y = j*DY;

            fprintf(file_V1, "%d %d %.3f %.3f %.3f\n", i, j, x, y, h_V[n]);
            fprintf(file_source, "%d %d %.3f %.3f %.3f\n", i, j, x, y, h_source[n]);
        }
    }
    //free GPU arrays
    free(h_V);
    free(h_source);

    //write data out
    printf("\nFinished!\n");
}
